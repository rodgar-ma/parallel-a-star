/* -------------------------------------------------------------------------- */
/* Project: I Curso de Computación Científica en Clusters                     */
/* Author:  Juan Fernández Peinador                                           */
/* Date:    Marzo de 2010                                                     */
/* Actualizado en Febrero 2021 para cuda 8.0: hipDeviceReset()		      */
/* -------------------------------------------------------------------------- */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

// includes, project
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>

// ayuda con los ejemplos
// These are CUDA Helper functions for initialization and error checking
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>



////////////////////////////////////////////////////////////////////////////////

// includes, kernels
#include "cuda_vectorReduce_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    float *vector_h, *reduce_h; // host data
    float *vector_d, *reduce_d; // device data
    size_t nBytes;

    // default values
    int n = 1;
    int bsx = 1;


 // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;


    //events
    float processing_time;
    hipEvent_t start_event, stop_event;	


    // process command line arguments
    n=getCmdLineArgumentInt(argc, (const char **) argv, (const char *) "n")?:n;
    bsx=getCmdLineArgumentInt(argc, (const char **) argv, (const char *) "bsx")?:bsx;

    nBytes = n * sizeof(float);

    // setup execution parameters
    dim3 grid( (n%bsx) ? (n/bsx)+1 : (n/bsx) );
    dim3 block(bsx);

    // allocate host memory
    vector_h = (float *) malloc(nBytes);
    for(int i = 0; i < n; i++)
        vector_h[i] = (float) 1.0;
    reduce_h = (float *) malloc(grid.x * sizeof(float));
    bzero(reduce_h, 1 * sizeof(float));
    
    // allocate device memory
    checkCudaErrors(hipMalloc((void **) &vector_d, nBytes));
    checkCudaErrors(hipMalloc((void **) &reduce_d, grid.x * sizeof(float)));

    // copy data from host memory to device memory
    checkCudaErrors(hipMemcpy(vector_d, vector_h, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(reduce_d, 0, grid.x * sizeof(float)));

    // execute the kernel
    printf("Running configuration: grid of %d blocks of %d threads (%d threads)\n", 
           grid.x, block.x, grid.x * block.x );


    //create events
    checkCudaErrors(hipEventCreate(&start_event,0));
    checkCudaErrors(hipEventCreate(&stop_event,0));
    
    //using events
    checkCudaErrors(hipEventRecord(start_event,0));


    vectorReduce<<<grid, block, block.x * sizeof(float)>>>(vector_d, reduce_d, n);
    
    // wait for thread completion
    hipDeviceSynchronize();





 // ///*using event*/        
    checkCudaErrors(hipEventRecord(stop_event, 0));        
    hipEventSynchronize(stop_event);   // block until the event is actually recorded        
    checkCudaErrors(hipEventElapsedTime(&processing_time, start_event, stop_event));        
    printf("Processing time: %f (ms)", processing_time);       


    checkCudaErrors(hipMemcpy(reduce_h, reduce_d, grid.x * sizeof(float), hipMemcpyDeviceToHost));



    //compute final stage
    for(int i = 1; i < grid.x; i++)
        reduce_h[0] += reduce_h[i];

    // check result
    assert(reduce_h[0] == (float) n);

    // free memory
    free(vector_h);
    free(reduce_h);
    checkCudaErrors(hipFree((void *) vector_d));
    checkCudaErrors(hipFree((void *) reduce_d));

    printf("\nTest PASSED\n");

    //    hipDeviceReset();

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }





}
